#include "hip/hip_runtime.h"
#include "../include/module.h"
#include "../include/rand.h"
#include "../include/timer.h"
#include <vector>
#include<algorithm>
#define BLOCK_DIM 256
#define TILE_WIDTH 32

/* error handling for CUDA API functions */
#define CHECK(call)                                                  \
    {                                                                \
        const hipError_t err = call;                                \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

/* check to kernel call */
#define CHECK_KERNELCALL()                                           \
    {                                                                \
        const hipError_t err = hipGetLastError();                  \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

float *input_data, *input_grad, *layer1_var1_data, *layer1_var1_grad, *layer1_var2_data, *layer1_var2_grad, *layer2_var1_data, *layer2_var1_grad, *output_data, *output_grad;
int max_dim_dropout = 0;
unsigned long long *rand1_gpu, *rand2_gpu;
int epoch = 0;
float *original_input_data;
int *src_index;

hipStream_t stream1;

// ################################################################################################################

/**
 * Dense matrix multiplication layer.
 */

Matmul::Matmul(Variable *a, Variable *b, Variable *c, int m, int n, int p) : a(a), b(b), c(c), m(m), n(n), p(p)
{
    CHECK(hipMalloc(&b_data, b->data.size() * sizeof(float)));
    CHECK(hipMalloc(&layer2_var1_data, c->data.size() * sizeof(float)));

    CHECK(hipMalloc(&b_grad, b->grad.size() * sizeof(float)));
    CHECK(hipMalloc(&layer2_var1_grad, c->grad.size() * sizeof(float)));
	
    if (m < 20000)
    {
        CHECK(hipMalloc(&b_sum, a->data.size() * b->data.size() * sizeof(float)));
    }

    CHECK(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
}

Matmul::~Matmul()
{
    CHECK(hipFree(b_data));
    CHECK(hipFree(b_grad));
    if (m < 20000)
    {
        CHECK(hipFree(b_sum));
    }
    CHECK(hipStreamDestroy(stream1));
}

__global__ void gpu_matmul_forward(float *a_data, float *b_data, float *c_data, const int m, const int n, const int p)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    if(idx >= m * p) return;
    int i = idx / p;
    int k = idx % p;

    __shared__ float local_vars[BLOCK_DIM];

    local_vars[thread_id] = 0;

    for (int j = 0; j < n; j++)
        local_vars[thread_id] += a_data[i * n + j] * b_data[j * p + k];

    c_data[i * p + k] = local_vars[thread_id];
}

__global__ void gpu_matmul_forward2(float *a, float *b, float *c, int a_rows, int a_columns, int b_rows, int b_columns, int c_rows, int c_columns)
{
    __shared__ float shared_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_b[TILE_WIDTH][TILE_WIDTH];

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    float c_val = 0.0;
    shared_a[threadIdx.y][threadIdx.x] = 0.0;
    shared_b[threadIdx.y][threadIdx.x] = 0.0;

    for (int ph = 0; ph < (((a_columns - 1) / TILE_WIDTH) + 1); ph++)
    {
        if (row < a_rows && (threadIdx.x + (ph * TILE_WIDTH)) < a_columns)
        {
            shared_a[threadIdx.y][threadIdx.x] = a[(row * a_columns) + threadIdx.x + (ph * TILE_WIDTH)];
        }
        else
        {
            shared_a[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (col < b_columns && (threadIdx.y + ph * TILE_WIDTH) < b_rows)
        {
            shared_b[threadIdx.y][threadIdx.x] = b[(threadIdx.y + ph * TILE_WIDTH) * b_columns + col];
        }
        else
        {
            shared_b[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; ++j)
        {
            c_val += shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
        }
    }
    if (row < c_rows && col < c_columns)
    {
        c[row * c_columns + col] = c_val;
    }
}

void Matmul::forward(bool training)
{
    timer_start(TMR_MATMUL_FW);

    if(epoch == 0 || !training)CHECK(hipMemcpy(b_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));

    if (m < 20000)
    {
        dim3 blocksPerGrid((m * p + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
        dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
        gpu_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_data, b_data, layer2_var1_data, m, n, p);
    }
    else
    {
        dim3 blocksPerGrid((p / TILE_WIDTH) + 1, (m / TILE_WIDTH) + 1, 1);
        dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH, 1);
        gpu_matmul_forward2<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_data, b_data, layer2_var1_data, m, n, n, p, m, p);
    }
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    timer_stop(TMR_MATMUL_FW);
}

__global__ void gpu_matmul_backward1(float *a_grad, float *b_data, float *c_grad, const int m, const int n, const int p)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    if(idx >= m * n) return;
    int i = idx / n;
    int j = idx % n;

    __shared__ float local_vars[BLOCK_DIM];

    local_vars[thread_id] = 0;

    for (int k = 0; k < p; k++)
    {
         local_vars[thread_id] += c_grad[i * p + k] * b_data[j * p + k];
    }

    a_grad[i * n + j] = local_vars[thread_id];
}

__global__ void gpu_matmul_backward2_copy(float *a_grad, float *a_data, float *c_grad, const int m, const int n, const int p, float *values)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int i = blockIdx.y;
    if(idx >= n * p || i >= m) return;
    int j = idx / p;
    int k = idx % p;
	
    values[i * n * n + j * p + k] = c_grad[i * p + k] * a_data[i * n + j];
}

__global__ void gpu_matmul_backward2(float *b_grad, const int n, const int p, float *values)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n * p) return;
    int j = idx / p;
    int k = idx % p;

    b_grad[j * p + k] = values[j * p + k];
}

__global__ void gpu_matmul_backward2_sum(float *values, const int dim, const int dim2, const int m, const int n, const int p){
    int pos = blockIdx.y;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n * p || pos >= dim2) return;
    int j = idx / (p);
    int k = idx % (p);
    if(dim % 2 == 0 || pos != int(dim / 2))
    {
     	values[pos * n * n + j * p + k] += values[(pos + dim2) * n * n + j * p + k];
    }
}

__global__ void gpu_matmul_backward3(float *b_grad, float *a_data, float *c_grad, const int m, const int n, const int p)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    if(idx >= n * p) return;
    int j = idx / p;
    int k = idx % p;

    __shared__ float local_vars[BLOCK_DIM];

    local_vars[thread_id] = 0;

    for (int i = 0; i < m; i++)
    {
         local_vars[thread_id] += c_grad[i * p + k] * a_data[i * n + j];
    }

    b_grad[j * p + k] = local_vars[thread_id];
}

void Matmul::backward()
{
    timer_start(TMR_MATMUL_BW);
    
    CHECK(hipMemcpyAsync(b_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice, stream1));

    dim3 blocksPerGrid1((m * n + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock1(BLOCK_DIM, 1, 1);
    gpu_matmul_backward1<<<blocksPerGrid1, threadsPerBlock1, 0, stream1>>>(layer1_var2_grad, b_data, layer2_var1_grad, m, n, p);
    CHECK_KERNELCALL();

    if (m < 20000)
    {
        int multiple32 = m + 32 - 1;
        multiple32 -= (multiple32 % 32);

        dim3 blocksPerGrid0((n * p + BLOCK_DIM - 1) / BLOCK_DIM, multiple32, 1);
        dim3 threadsPerBlock0(BLOCK_DIM, 1, 1);
        gpu_matmul_backward2_copy<<<blocksPerGrid0, threadsPerBlock0>>>(layer1_var2_grad, layer1_var2_data, layer2_var1_grad, m, n, p, b_sum);
        CHECK_KERNELCALL();

        dim3 blocksPerGridSum((n * p + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
        dim3 threadsPerBlockSum(BLOCK_DIM, 1, 1);

        int dim = m;
        int dim2 = m;

        for (int x = 0; x < ceil(log2(m)); x++)
        {
            dim2 = ceil(dim2 / 2.0);
            multiple32 = dim2 + 32 - 1;
            multiple32 -= (dim2 % 32);
            blocksPerGridSum.y = multiple32;
            gpu_matmul_backward2_sum<<<blocksPerGridSum, threadsPerBlockSum>>>(b_sum, dim, dim2, m, n, p);
            CHECK_KERNELCALL();
            dim = dim2;
        }

        dim3 blocksPerGrid2((n * p + BLOCK_DIM - 1), 1, 1);
        dim3 threadsPerBlock2(BLOCK_DIM, 1, 1);
        gpu_matmul_backward2<<<blocksPerGrid2, threadsPerBlock2>>>(b_grad, n, p, b_sum);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
    }
    else
    {
        dim3 blocksPerGrid2((n * p + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
        dim3 threadsPerBlock2(BLOCK_DIM, 1, 1);
        gpu_matmul_backward3<<<blocksPerGrid2, threadsPerBlock2>>>(b_grad, layer1_var2_data, layer2_var1_grad, m, n, p);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
    }

    CHECK(hipMemcpy(&a->grad[0], layer1_var2_grad, sizeof(float) * a->grad.size(), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&b->grad[0], b_grad, sizeof(float) * b->grad.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_MATMUL_BW);
}

// ################################################################################################################

int *i_index;

/**
 * A sparse matrix multiplication layer.
 */
SparseMatmul::SparseMatmul(Variable *a, Variable *b, Variable *c, SparseIndex *sp, int m, int n, int p) : a(a), b(b), c(c), sp(sp), m(m), n(n), p(p)
{
    CHECK(hipMalloc(&b_data, b->data.size() * sizeof(float)));
    CHECK(hipMalloc(&layer1_var1_data, c->data.size() * sizeof(float)));

    CHECK(hipMalloc(&b_grad, b->grad.size() * sizeof(float)));
    CHECK(hipMalloc(&layer1_var1_grad, c->grad.size() * sizeof(float)));

    CHECK(hipMalloc(&sp_indptr, sp->indptr.size() * sizeof(float)));
    CHECK(hipMalloc(&sp_indices, sp->indices.size() * sizeof(float)));

    CHECK(hipMemcpy(sp_indptr, &(sp->indptr[0]), sizeof(int) * sp->indptr.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(sp_indices, &(sp->indices[0]), sizeof(int) * sp->indices.size(), hipMemcpyHostToDevice));
	
	std::vector<std::pair<int, int> > i_length;
	for(int i = 0; i < sp->indptr.size() - 1; i++){
		i_length.push_back({sp->indptr[i + 1] - sp->indptr[i], i});
	}
	std::sort(i_length.begin(), i_length.end());
	std::reverse(i_length.begin(), i_length.end());
	int *i_length_index;
	i_length_index = (int *)malloc((sp->indptr.size() - 1) * sizeof(int));
	for(int i = 0; i < sp->indptr.size() - 1; i++){
		i_length_index[i] = i_length[i].second;
	}
	CHECK(hipMalloc(&i_index, (sp->indptr.size() - 1) * sizeof(int)));
	CHECK(hipMemcpy(i_index, i_length_index, (sp->indptr.size() - 1) * sizeof(int), hipMemcpyHostToDevice));
}

SparseMatmul::~SparseMatmul()
{
    CHECK(hipFree(b_data));
    CHECK(hipFree(b_grad));
    CHECK(hipFree(sp_indptr));
    CHECK(hipFree(sp_indices));
}

__global__ void gpu_sparse_matmul_forward(int *i_index, float *a_data, float *b_data, float *c_data, int *sp_indptr, int *sp_indices, const int p, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    if(idx >= idx_max) return;
    int ind_i = idx / p;
	int i = i_index[ind_i];
    int k = idx % p;

    __shared__ float local_vars[BLOCK_DIM];

    local_vars[thread_id] = 0;

    for (int jj = sp_indptr[i]; jj < sp_indptr[i + 1]; jj++)
    {
        int j = sp_indices[jj];
        local_vars[thread_id] += a_data[jj] * b_data[j * p + k];
    }

    c_data[i * p + k] = local_vars[thread_id];
}

void SparseMatmul::forward(bool training)
{
    timer_start(TMR_SPMATMUL_FW);

    CHECK(hipMemcpy(b_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(((sp->indptr.size() - 1) * p + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    gpu_sparse_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(i_index, input_data, b_data, layer1_var1_data, sp_indptr, sp_indices, p, (sp->indptr.size() - 1) * p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    timer_stop(TMR_SPMATMUL_FW);
}

__global__ void gpu_sparse_matmul_backward(int *i_index, float *a_data, float *b_grad, float *c_grad, int *sp_indptr, int *sp_indices, const int p, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= idx_max) return;
    int i = idx / p;
    int k = idx % p;

    for (int jj = sp_indptr[i]; jj < sp_indptr[i + 1]; jj++)
    {
        int j = sp_indices[jj];
        atomicAdd(&b_grad[j * p + k], c_grad[i * p + k] * a_data[jj]);
    }
}

void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);

    CHECK(hipMemset(b_grad, 0, sizeof(float) * b->grad.size()));

    dim3 blocksPerGrid(((sp->indptr.size() - 1) * p + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    gpu_sparse_matmul_backward<<<blocksPerGrid, threadsPerBlock>>>(i_index, input_data, b_grad, layer1_var1_grad, sp_indptr, sp_indices, p, sp->indptr.size() * p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&b->grad[0], b_grad, sizeof(float) * b->grad.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_SPMATMUL_BW);
}

// ################################################################################################################

/**
 * A specialized sparse matrix multiplication for graphs.
 */
GraphSum::GraphSum(Variable *in, Variable *out, SparseIndex *graph, int dim, bool isFirst) : in(in), out(out), graph(graph), dim(dim), isFirst(isFirst)
{
    if (isFirst)
    {
        CHECK(hipMalloc(&layer1_var2_data, out->data.size() * sizeof(float)));
        CHECK(hipMalloc(&layer1_var2_grad, out->grad.size() * sizeof(float)));
		
		std::vector<std::pair<int, int> > src_length;
		for(int i = 0; i < graph->indptr.size() - 1; i++){
			src_length.push_back({graph->indptr[i + 1] - graph->indptr[i], i});
		}
		std::sort(src_length.begin(), src_length.end());
		std::reverse(src_length.begin(), src_length.end());
		int *src_length_index;
		src_length_index = (int *)malloc((graph->indptr.size() - 1) * sizeof(int));
		for(int i = 0; i < graph->indptr.size() - 1; i++){
			src_length_index[i] = src_length[i].second;
		}
		CHECK(hipMalloc(&src_index, (graph->indptr.size() - 1) * sizeof(int)));
		CHECK(hipMemcpy(src_index, src_length_index, (graph->indptr.size() - 1) * sizeof(int), hipMemcpyHostToDevice));
    }
    else
    {
        CHECK(hipMalloc(&output_data, out->data.size() * sizeof(float)));
        CHECK(hipMalloc(&output_grad, out->grad.size() * sizeof(float)));
    }
    CHECK(hipMalloc(&graph_indptr, graph->indptr.size() * sizeof(int)));
    CHECK(hipMalloc(&graph_indices, graph->indices.size() * sizeof(int)));

    CHECK(hipMemcpy(graph_indptr, &(graph->indptr[0]), sizeof(int) * graph->indptr.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(graph_indices, &(graph->indices[0]), sizeof(int) * graph->indices.size(), hipMemcpyHostToDevice));
	
    max_diff = 0;
    for(int i = 1; i < graph->indptr.size(); i++)
    {
    	max_diff = max(max_diff, graph->indptr[i] - graph->indptr[i - 1]);
    }
}

GraphSum::~GraphSum()
{
    CHECK(hipFree(graph_indptr));
    CHECK(hipFree(graph_indices));
}

__global__ void gpu_graph_sum_forward_zero(float *in_data, float *out_data, int *graph_indptr, int *graph_indices, const int dim, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= idx_max) return;
    int src = idx / dim;
    int j = idx % dim;

    out_data[src * dim + j] = 0;
}

__global__ void gpu_graph_sum_forward(float *in_data, float *out_data, int *graph_indptr, int *graph_indices, const int dim, const int length, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx > idx_max) return;
    int src = idx / dim;
    int j = idx % dim;
    int delta_i = blockIdx.y;

    float sum = 0;
	
    for (int i = graph_indptr[src] + delta_i; i < graph_indptr[src + 1]; i += length)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf((graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
	    sum += coef * in_data[dst * dim + j];
    }
	atomicAdd(&out_data[src * dim + j], sum);
}

__global__ void gpu_graph_sum_forward2(int *src_index, float *in_data, float *out_data, int *graph_indptr, int *graph_indices, const int dim, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx > idx_max) return;
    int ind_src = idx / dim;
	int src = src_index[ind_src];
    int j = idx % dim;

    float sum = 0;
	
    for (int i = graph_indptr[src]; i < graph_indptr[src + 1]; i++)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf((graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
	    sum += coef * in_data[dst * dim + j];
    }
	out_data[src * dim + j] = sum;
}

void GraphSum::forward(bool training)
{
    timer_start(TMR_GRAPHSUM_FW);

	if (graph->indptr.size() - 1 < 5000) {
		dim3 blocksPerGrid0(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
		dim3 threadsPerBlock0(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_forward_zero<<<blocksPerGrid0, threadsPerBlock0>>>(layer1_var1_data, layer1_var2_data, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		if (!isFirst)
			gpu_graph_sum_forward_zero<<<blocksPerGrid0, threadsPerBlock0>>>(layer2_var1_data, output_data, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		CHECK_KERNELCALL();

		dim3 blocksPerGrid(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, sqrt(max_diff), 1);
		dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_forward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var1_data, layer1_var2_data, graph_indptr, graph_indices, dim, sqrt(max_diff), (graph->indptr.size() - 1) * dim);
		else
			gpu_graph_sum_forward<<<blocksPerGrid, threadsPerBlock>>>(layer2_var1_data, output_data, graph_indptr, graph_indices, dim, sqrt(max_diff), (graph->indptr.size() - 1) * dim);
		CHECK_KERNELCALL();
		CHECK(hipDeviceSynchronize());
	} else {
	    dim3 blocksPerGrid(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
		dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_forward2<<<blocksPerGrid, threadsPerBlock>>>(src_index, layer1_var1_data, layer1_var2_data, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		else
			gpu_graph_sum_forward2<<<blocksPerGrid, threadsPerBlock>>>(src_index, layer2_var1_data, output_data, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		CHECK_KERNELCALL();
		CHECK(hipDeviceSynchronize());
	}
	if (isFirst)
	{
		CHECK(hipMemcpy(&out->data[0], layer1_var2_data, sizeof(float) * out->data.size(), hipMemcpyDeviceToHost));
	}
	else
	{
		CHECK(hipMemcpy(&out->data[0], output_data, sizeof(float) * out->data.size(), hipMemcpyDeviceToHost));
	}
    timer_stop(TMR_GRAPHSUM_FW);
}

__global__ void gpu_graph_sum_backward_zero(float *in_grad, float *out_grad, int *graph_indptr, int *graph_indices, const int dim, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= idx_max) return;
    int src = idx / dim;
    int j = idx % dim;

    in_grad[src * dim + j] = 0;
}

__global__ void gpu_graph_sum_backward(float *in_grad, float *out_grad, int *graph_indptr, int *graph_indices, const int dim, const int length, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx > idx_max) return;
    int src = idx / dim;
    int j = idx % dim;
    int delta_i = blockIdx.y;
	
    float sum = 0;
	
    for (int i = graph_indptr[src] + delta_i; i < graph_indptr[src + 1]; i += length)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf(
                               (graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
        sum += coef * out_grad[dst * dim + j];
    }
    atomicAdd(&in_grad[src * dim + j], sum);
}

__global__ void gpu_graph_sum_backward2(int *src_index, float *in_grad, float *out_grad, int *graph_indptr, int *graph_indices, const int dim, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx > idx_max) return;
    int ind_src = idx / dim;
	int src = src_index[ind_src];
    int j = idx % dim;
	
    float sum = 0;
	
    for (int i = graph_indptr[src]; i < graph_indptr[src + 1]; i++)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf(
                               (graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
        sum += coef * out_grad[dst * dim + j];
    }
    in_grad[src * dim + j] = sum;
}

void GraphSum::backward()
{
    timer_start(TMR_GRAPHSUM_BW);

	if (graph->indptr.size() - 1 < 5000) {
		dim3 blocksPerGrid0(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
		dim3 threadsPerBlock0(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_backward_zero<<<blocksPerGrid0, threadsPerBlock0>>>(layer1_var1_grad, layer1_var2_grad, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		if (!isFirst)
			gpu_graph_sum_backward_zero<<<blocksPerGrid0, threadsPerBlock0>>>(layer2_var1_grad, output_grad, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);

		dim3 blocksPerGrid(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, sqrt(max_diff), 1);
		dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_backward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var1_grad, layer1_var2_grad, graph_indptr, graph_indices, dim, sqrt(max_diff), (graph->indptr.size() - 1) * dim);
		else
			gpu_graph_sum_backward<<<blocksPerGrid, threadsPerBlock>>>(layer2_var1_grad, output_grad, graph_indptr, graph_indices, dim, sqrt(max_diff), (graph->indptr.size() - 1) * dim);
		CHECK_KERNELCALL();
		CHECK(hipDeviceSynchronize());
	} else {
	    dim3 blocksPerGrid(((graph->indptr.size() - 1) * dim + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
		dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
		if (isFirst)
			gpu_graph_sum_backward2<<<blocksPerGrid, threadsPerBlock>>>(src_index, layer1_var1_grad, layer1_var2_grad, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		else
			gpu_graph_sum_backward2<<<blocksPerGrid, threadsPerBlock>>>(src_index, layer2_var1_grad, output_grad, graph_indptr, graph_indices, dim, (graph->indptr.size() - 1) * dim);
		CHECK_KERNELCALL();
		CHECK(hipDeviceSynchronize());
	}
    timer_stop(TMR_GRAPHSUM_BW);
}

// ################################################################################################################

/**
 * Each predicted class probability is compared to the actual class desired and a loss is computed to penalize the proabability based on how far it is with respect to the actual expected value.
 * Also called logaritmic loss. 
*/
CrossEntropyLoss::CrossEntropyLoss(Variable *logits, int *truth_training, int *truth_validation, int *truth_testing, float *loss, int num_classes) :
        logits(logits), truth_training(truth_training), truth_validation(truth_validation), truth_testing(truth_testing), loss(loss), num_classes(num_classes) 
{
    CHECK(hipMalloc(&count_gpu, sizeof(int)));
    CHECK(hipMalloc(&total_loss_gpu, sizeof(float)));

    CHECK(hipMalloc(&truth_training_gpu, sizeof(int) * (logits->data.size() / num_classes)));
    CHECK(hipMalloc(&truth_validation_gpu, sizeof(int) * (logits->data.size() / num_classes)));
    CHECK(hipMalloc(&truth_testing_gpu, sizeof(int) * (logits->data.size() / num_classes)));
    CHECK(hipMemcpy(truth_training_gpu, truth_training, sizeof(int) * (logits->data.size() / num_classes), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(truth_validation_gpu, truth_validation, sizeof(int) * (logits->data.size() / num_classes), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(truth_testing_gpu, truth_testing, sizeof(int) * (logits->data.size() / num_classes), hipMemcpyHostToDevice));
}

CrossEntropyLoss::~CrossEntropyLoss()
{
    CHECK(hipFree(input_data));
    CHECK(hipFree(input_grad));
    CHECK(hipFree(layer1_var1_data));
    CHECK(hipFree(layer1_var1_grad));
    CHECK(hipFree(layer1_var2_data));
    CHECK(hipFree(layer1_var2_grad));
    CHECK(hipFree(layer2_var1_data));
    CHECK(hipFree(layer2_var1_grad));
    CHECK(hipFree(output_data));
    CHECK(hipFree(output_grad));
    CHECK(hipFree(original_input_data));
    CHECK(hipFree(rand1_gpu));
	CHECK(hipFree(rand2_gpu));
	CHECK(hipFree(src_index));
	CHECK(hipFree(i_index));
}

__global__ void gpu_cross_entropy_loss_forward1(int *truth, int *count, float *logits_data, float *total_loss, float *logits_grad, const bool training, const int idx_max, const int num_classes){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i >= idx_max || truth[i] < 0) return;

    atomicAdd(count, 1);

    float *logit = &logits_data[i * num_classes];
    float max_logit = -1e30, sum_exp = 0;
    for (int j = 0; j < num_classes; j++)
        max_logit = fmax(max_logit, logit[j]);
    for (int j = 0; j < num_classes; j++)
    {
        logit[j] -= max_logit;
        sum_exp += expf(logit[j]);
    }
    atomicAdd(total_loss, logf(sum_exp) - logit[truth[i]]);

    if (training)
    {
        for (int j = 0; j < num_classes; j++)
        {
            float prob = expf(logit[j]) / sum_exp;
            logits_grad[i * num_classes + j] = prob;
        }
        logits_grad[i * num_classes + truth[i]] -= 1.0;
    }
}

__global__ void gpu_cross_entropy_loss_forward2(float *logits_grad, const int count, const int idx_max)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= idx_max) return;

    logits_grad[i] /= count;
}

void CrossEntropyLoss::forward(bool training) {
    
    timer_start(TMR_LOSS_FW);
    float total_loss = 0;
    int count = 0;
    CHECK(hipMemset(count_gpu, 0, sizeof(int)));
    CHECK(hipMemset(total_loss_gpu, 0.0, sizeof(float)));

    dim3 blocksPerGrid(((logits->data.size() / num_classes) + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    if (training)
    {
        gpu_cross_entropy_loss_forward1<<<blocksPerGrid, threadsPerBlock>>>(truth_training_gpu, count_gpu, output_data, total_loss_gpu, output_grad, training, (logits->data.size() / num_classes), num_classes);
    }
    else
    {
        if (epoch < 100)
        {
            gpu_cross_entropy_loss_forward1<<<blocksPerGrid, threadsPerBlock>>>(truth_validation_gpu, count_gpu, output_data, total_loss_gpu, output_grad, training, (logits->data.size() / num_classes), num_classes);
            epoch++;
        }
        else
        {
            gpu_cross_entropy_loss_forward1<<<blocksPerGrid, threadsPerBlock>>>(truth_testing_gpu, count_gpu, output_data, total_loss_gpu, output_grad, training, (logits->data.size() / num_classes), num_classes);
        }
    }
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&total_loss, total_loss_gpu, sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&count, count_gpu, sizeof(int), hipMemcpyDeviceToHost));

    *loss = total_loss / count;
    if (training)
    {
        blocksPerGrid.x = (logits->grad.size() + BLOCK_DIM - 1) / BLOCK_DIM;
        gpu_cross_entropy_loss_forward2<<<blocksPerGrid, threadsPerBlock>>>(output_grad, count, logits->grad.size());
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
    }

    timer_stop(TMR_LOSS_FW);
}

void CrossEntropyLoss::backward() {
}

// ################################################################################################################

/**
 * Rectified Linear Unit activation function.
 * If input is negative it will output 0.
 */
ReLU::ReLU(Variable *in)
{
    this->in = in;
    mask = new bool[in->data.size()];
	
    CHECK(hipMalloc(&mask_gpu, in->data.size() * sizeof(bool)));
}

ReLU::~ReLU()
{
    delete[] mask;
    CHECK(hipFree(mask_gpu));
}

__global__ void gpu_relu_forward(float *in_data, bool *mask, const bool training, const int idx_max){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= idx_max) return;
	
    bool keep = in_data[i] > 0;
    if (training)
        mask[i] = keep;
    if (!keep)
        in_data[i] = 0;	
}

void ReLU::forward(bool training)
{
    timer_start(TMR_RELU_FW);
	
    dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    gpu_relu_forward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_data, mask_gpu, training, in->data.size());
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
	
    CHECK(hipMemcpy(mask, mask_gpu, in->data.size() * sizeof(bool), hipMemcpyDeviceToHost));
	
    timer_stop(TMR_RELU_FW);
}

__global__ void gpu_relu_backward(float *in_grad, bool *mask, const int idx_max){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= idx_max) return;
	
    if (!mask[i])
        in_grad[i] = 0;
}

void ReLU::backward()
{
    timer_start(TMR_RELU_BW);
	
    CHECK(hipMemcpy(mask_gpu, mask, in->data.size() * sizeof(bool), hipMemcpyHostToDevice));
	
    dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    gpu_relu_backward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_grad, mask_gpu, in->data.size());
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    timer_stop(TMR_RELU_BW);
}

// ################################################################################################################

/**
 * The dropout layer randomly sets input units to 0 with a frequency of P at each step during training time to prevent overfitting.
 * Inputs that are not set to 0 are scaled up by 1/(1-P).
 */
Dropout::Dropout(Variable *in, float p, bool isFirst, std::string input_name) : isFirst(isFirst), input_name(input_name)
{
    this->in = in;
    this->p = p;
    if (!in->grad.empty())
    {
        mask = new int[in->data.size()];
        CHECK(hipMalloc(&mask_gpu, in->data.size() * sizeof(int)));
    }
    else
    {
        mask = nullptr;
    }

    if (isFirst)
    {
        CHECK(hipMalloc(&input_data, in->data.size() * sizeof(float)));
        CHECK(hipMalloc(&input_grad, in->grad.size() * sizeof(float)));
        max_dim_dropout = in->data.size();
        if (input_name != "citeseer") 
        {
            CHECK(hipMalloc(&original_input_data, in->data.size() * sizeof(float)));
            CHECK(hipMemcpy(original_input_data, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
        }
    }
    else
    {
        srand(time(NULL));
        if (in->data.size() > max_dim_dropout)
            max_dim_dropout = in->data.size();
		unsigned long long *rand1, *rand2;
        rand1 = (unsigned long long *)malloc(max_dim_dropout * sizeof(unsigned long long));
        rand2 = (unsigned long long *)malloc(max_dim_dropout * sizeof(unsigned long long));
        for (int i = 0; i < max_dim_dropout; i++)
        {
            rand1[i] = rand();
            rand2[i] = rand();
            while (rand1[i] == 0 || rand2[i] == 0)
            {
                rand1[i] = rand();
                rand2[i] = rand();
            }
        }
        CHECK(hipMalloc(&rand1_gpu, max_dim_dropout * sizeof(unsigned long long)));
        CHECK(hipMalloc(&rand2_gpu, max_dim_dropout * sizeof(unsigned long long)));
        CHECK(hipMemcpy(rand1_gpu, rand1, max_dim_dropout * sizeof(unsigned long long), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(rand2_gpu, rand2, max_dim_dropout * sizeof(unsigned long long), hipMemcpyHostToDevice));
    }
}

Dropout::~Dropout()
{
    if (mask)
    {
        delete[] mask;
        CHECK(hipFree(mask_gpu));
    }
}

__global__ void gpu_set_original_input(float *in_data, float *original_input_data, const int idx_max)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= idx_max) return;
	
    in_data[i] = original_input_data[i];
}

__global__ void gpu_dropout_forward(float *in_data, int *mask, const bool isMask, const int threshold, const int scale, const int idx_max, unsigned long long *rand1, unsigned long long *rand2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= idx_max) return;

    unsigned long long t = rand1[i];
    unsigned long long const s = rand2[i];
    assert(t && s);
    rand1[i] = s;
    t ^= t << 23;		// a
    t ^= t >> 17;		// b
    t ^= s ^ (s >> 26);	// c
    rand2[i] = t;
    unsigned int res = (t + s) & 0x7fffffff;
    int rand = (int)res;

    in_data[i] *= (rand >= threshold) ? scale : 0;
    if (isMask)
        mask[i] = (rand >= threshold);
}

void Dropout::forward(bool training)
{
    if (!training)
    {
        if (isFirst)
        {
            if (input_name != "citeseer") {
                dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
                dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
                gpu_set_original_input<<<blocksPerGrid, threadsPerBlock>>>(input_data, original_input_data, in->data.size());
                CHECK_KERNELCALL();
            }
            else {
                CHECK(hipMemcpy(input_data, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
            }
	    }
        return;
    }
    timer_start(TMR_DROPOUT_FW);
    const int threshold = int(p * MY_RAND_MAX);
    float scale = 1 / (1 - p);

    if (isFirst)
    {
        if (input_name != "citeseer") {
            dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
            dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
            gpu_set_original_input<<<blocksPerGrid, threadsPerBlock>>>(input_data, original_input_data, in->data.size());
            CHECK_KERNELCALL();
        } else {
            CHECK(hipMemcpy(input_data, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
        }
    }

    bool isMask = false;
    if (mask)
    {
        CHECK(hipMemcpy(mask_gpu, mask, sizeof(int) * in->data.size(), hipMemcpyHostToDevice));
        isMask = true;
    }
    
    dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    if (isFirst)
        gpu_dropout_forward<<<blocksPerGrid, threadsPerBlock>>>(input_data, mask_gpu, isMask, threshold, scale, in->data.size(), rand1_gpu, rand2_gpu);
    else
        gpu_dropout_forward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_data, mask_gpu, isMask, threshold, scale, in->data.size(), rand1_gpu, rand2_gpu);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    timer_stop(TMR_DROPOUT_FW);
}

__global__ void gpu_dropout_backward(float *in_grad, int *mask, const int scale, const int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= idx_max) return;

    in_grad[idx] *= mask[idx] ? scale : 0;
}

void Dropout::backward()
{
    if (!mask)
        return;
    
    timer_start(TMR_DROPOUT_BW);
    float scale = 1 / (1 - p);

    dim3 blocksPerGrid((in->data.size() + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    if (isFirst)
    {
        gpu_dropout_backward<<<blocksPerGrid, threadsPerBlock>>>(input_grad, mask_gpu, scale, in->data.size());
    }
    else
    {
        gpu_dropout_backward<<<blocksPerGrid, threadsPerBlock>>>(layer1_var2_grad, mask_gpu, scale, in->data.size());
    }
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    timer_stop(TMR_DROPOUT_BW);
}

// ################################################################################################################
