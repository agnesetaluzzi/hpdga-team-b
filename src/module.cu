#include "hip/hip_runtime.h"
#include "../include/module.h"
#include "../include/rand.h"
#include "../include/timer.h"
#include <vector>

/* error handling for CUDA API functions */
#define CHECK(call)                                                  \
    {                                                                \
        const hipError_t err = call;                                \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

/* Check to kernel call */
#define CHECK_KERNELCALL()                                           \
    {                                                                \
        const hipError_t err = hipGetLastError();                  \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

// ################################################################################################################

float *a_data, *b_data, *c_data;
float *a_grad, *b_grad, *c_grad;

/**
 * Dense matrix multiplication layer.
 */
Matmul::Matmul(Variable *a, Variable *b, Variable *c, int m, int n, int p) : a(a), b(b), c(c), m(m), n(n), p(p)
{
    CHECK(hipMalloc(&a_data, a->data.size() * sizeof(float)));
    CHECK(hipMalloc(&b_data, b->data.size() * sizeof(float)));
    CHECK(hipMalloc(&c_data, c->data.size() * sizeof(float)));

    CHECK(hipMalloc(&a_grad, a->grad.size() * sizeof(float)));
    CHECK(hipMalloc(&b_grad, b->grad.size() * sizeof(float)));
    CHECK(hipMalloc(&c_grad, c->grad.size() * sizeof(float)));
}

__global__ void gpu_matmul_forward(float *a_gpu, float *b_gpu, float *c_gpu, const int m, const int n, const int p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    c_gpu[i * p + k] = 0;

    for (int j = 0; j < n; j++)
        c_gpu[i * p + k] += a_gpu[i * n + j] * b_gpu[j * p + k];
}

void Matmul::forward(bool training)
{
    timer_start(TMR_MATMUL_FW);

    CHECK(hipMemcpy(a_data, &(a->data[0]), sizeof(float) * a->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(c_data, &(c->data[0]), sizeof(float) * c->data.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(m, 1, 1);
    dim3 threadsPerBlock(p, 1, 1);
    gpu_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(a_data, b_data, c_data, m, n, p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&c->data[0], c_data, sizeof(float) * c->data.size(), hipMemcpyDeviceToHost));
    timer_stop(TMR_MATMUL_FW);
}

__global__ void gpu_matmul_backward1(float *a_grad, float *a_data, float *b_data, float *b_grad, float *c_grad, const int m, const int n, const int p)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    a_grad[i * n + j] = 0;

    for (int k = 0; k < p; k++)
    {
        a_grad[i * n + j] += c_grad[i * p + k] * b_data[j * p + k];
    }
}

__global__ void gpu_matmul_backward2(float *a_grad, float *a_data, float *b_data, float *b_grad, float *c_grad, const int m, const int n, const int p)
{
    int j = blockIdx.x;
    int k = threadIdx.x;

    b_grad[j * p + k] = 0;

    for (int i = 0; i < m; i++)
    {
        b_grad[j * p + k] += c_grad[i * p + k] * a_data[i * n + j];
    }
}

void Matmul::backward()
{
    timer_start(TMR_MATMUL_BW);
    CHECK(hipMemcpy(a_grad, &(a->grad[0]), sizeof(float) * a->grad.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(a_data, &(a->data[0]), sizeof(float) * a->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_grad, &(b->grad[0]), sizeof(float) * b->grad.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(c_grad, &(c->grad[0]), sizeof(float) * c->grad.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid1(m, 1, 1);
    dim3 threadsPerBlock1(n, 1, 1);
    gpu_matmul_backward1<<<blocksPerGrid1, threadsPerBlock1>>>(a_grad, a_data, b_data, b_grad, c_grad, m, n, p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    dim3 blocksPerGrid2(n, 1, 1);
    dim3 threadsPerBlock2(p, 1, 1);
    gpu_matmul_backward2<<<blocksPerGrid2, threadsPerBlock2>>>(a_grad, a_data, b_data, b_grad, c_grad, m, n, p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&a->grad[0], a_grad, sizeof(float) * a->grad.size(), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&b->grad[0], b_grad, sizeof(float) * b->grad.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_MATMUL_BW);
}

// ################################################################################################################

float *a2_data, *b2_data, *c2_data;
float *a2_grad, *b2_grad, *c2_grad;
int *sp_indptr, *sp_indices;

/**
 * A sparse matrix multiplication layer.
 */

SparseMatmul::SparseMatmul(Variable *a, Variable *b, Variable *c, SparseIndex *sp, int m, int n, int p) : a(a), b(b), c(c), sp(sp), m(m), n(n), p(p)
{
    hipMalloc(&a2_data, a->data.size() * sizeof(float));
    hipMalloc(&b2_data, b->data.size() * sizeof(float));
    hipMalloc(&c2_data, c->data.size() * sizeof(float));

    hipMalloc(&a2_grad, a->grad.size() * sizeof(float));
    hipMalloc(&b2_grad, b->grad.size() * sizeof(float));
    hipMalloc(&c2_grad, c->grad.size() * sizeof(float));

    hipMalloc(&sp_indptr, sp->indptr.size() * sizeof(float));
    hipMalloc(&sp_indices, sp->indices.size() * sizeof(float));
}

__global__ void gpu_sparse_matmul_forward(float *a_data, float *b_data, float *c_data, int *sp_indptr, int *sp_indices, const int p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    c_data[i * p + k] = 0;

    for (int jj = sp_indptr[i]; jj < sp_indptr[i + 1]; jj++)
    {
        int j = sp_indices[jj];
        c_data[i * p + k] += a_data[jj] * b_data[j * p + k];
    }
}

void SparseMatmul::forward(bool training)
{
    timer_start(TMR_SPMATMUL_FW);

    CHECK(hipMemcpy(sp_indptr, &(sp->indptr[0]), sizeof(int) * sp->indptr.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(sp_indices, &(sp->indices[0]), sizeof(int) * sp->indices.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(a2_data, &(a->data[0]), sizeof(float) * a->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b2_data, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(c2_data, &(c->data[0]), sizeof(float) * c->data.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(sp->indptr.size() - 1, 1, 1);
    dim3 threadsPerBlock(p, 1, 1);
    gpu_sparse_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(a2_data, b2_data, c2_data, sp_indptr, sp_indices, p);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&c->data[0], c2_data, sizeof(float) * c->data.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_SPMATMUL_FW);
}

__global__ void gpu_sparse_matmul_backward(float *a_data, float *b_grad, float *c_grad, int *sp_indptr, int *sp_indices, const int p, const int sp_indptr_size)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    for (int jj = sp_indptr[i]; jj < sp_indptr[i + 1]; jj++)
    {
        int j = sp_indices[jj];
        atomicAdd(&b_grad[j * p + k], c_grad[i * p + k] * a_data[jj]);
    }
}

void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);

    CHECK(hipMemcpy(sp_indptr, &(sp->indptr[0]), sizeof(int) * sp->indptr.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(sp_indices, &(sp->indices[0]), sizeof(int) * sp->indices.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(a2_data, &(a->data[0]), sizeof(float) * a->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemset(b2_grad, 0, sizeof(float) * b->grad.size()));
    CHECK(hipMemcpy(c2_grad, &(c->grad[0]), sizeof(float) * c->grad.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(sp->indptr.size() - 1, 1, 1);
    dim3 threadsPerBlock(p, 1, 1);
    gpu_sparse_matmul_backward<<<blocksPerGrid, threadsPerBlock>>>(a2_data, b2_grad, c2_grad, sp_indptr, sp_indices, p, sp->indptr.size());
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(&b->grad[0], b2_grad, sizeof(float) * b->grad.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_SPMATMUL_BW);
}

// ################################################################################################################

float *in_data, *out_data;
float *in_grad, *out_grad;
int *graph_indptr, *graph_indices;

float *in_data2, *out_data2;
float *in_grad2, *out_grad2;
int *graph_indptr2, *graph_indices2;

int call_forward = 0;
int call_backward = 1;

/**
 * A specialized sparse matrix multiplication for graphs.
 */
GraphSum::GraphSum(Variable *in, Variable *out, SparseIndex *graph, int dim) : in(in), out(out), graph(graph), dim(dim)
{
    if(call_forward == 0)
    {
        hipMalloc(&in_data, in->data.size() * sizeof(float));
        hipMalloc(&out_data, out->data.size() * sizeof(float));
        hipMalloc(&in_grad, in->grad.size() * sizeof(float));
        hipMalloc(&out_grad, out->grad.size() * sizeof(float));
        hipMalloc(&graph_indptr, graph->indptr.size() * sizeof(int));
        hipMalloc(&graph_indices, graph->indices.size() * sizeof(int));
    }
    else{
        hipMalloc(&in_data2, in->data.size() * sizeof(float));
        hipMalloc(&out_data2, out->data.size() * sizeof(float));
        hipMalloc(&in_grad2, in->grad.size() * sizeof(float));
        hipMalloc(&out_grad2, out->grad.size() * sizeof(float));
        hipMalloc(&graph_indptr2, graph->indptr.size() * sizeof(int));
        hipMalloc(&graph_indices2, graph->indices.size() * sizeof(int));
    }
    call_forward += 1;
    call_forward %= 2;
}

__global__ void gpu_graph_sum_forward(float *in_data, float *out_data, int *graph_indptr, int *graph_indices, const int dim)
{
    int src = blockIdx.x;
    int j = threadIdx.x;

    out_data[src * dim + j] = 0;

    for (int i = graph_indptr[src]; i < graph_indptr[src + 1]; i++)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf((graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
        out_data[src * dim + j] += coef * in_data[dst * dim + j];
    }
}

void GraphSum::forward(bool training)
{
    timer_start(TMR_GRAPHSUM_FW);
    if(call_forward == 0)
    {
        CHECK(hipMemcpy(graph_indptr, &(graph->indptr[0]), sizeof(int) * graph->indptr.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(graph_indices, &(graph->indices[0]), sizeof(int) * graph->indices.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(in_data, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(out_data, &(out->data[0]), sizeof(float) * out->data.size(), hipMemcpyHostToDevice));

        dim3 blocksPerGrid(graph->indptr.size() - 1, 1, 1);
        dim3 threadsPerBlock(dim, 1, 1);
        gpu_graph_sum_forward<<<blocksPerGrid, threadsPerBlock>>>(in_data, out_data, graph_indptr, graph_indices, dim);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(&out->data[0], out_data, sizeof(float) * out->data.size(), hipMemcpyDeviceToHost));
    } else
    {
        CHECK(hipMemcpy(graph_indptr2, &(graph->indptr[0]), sizeof(int) * graph->indptr.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(graph_indices2, &(graph->indices[0]), sizeof(int) * graph->indices.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(in_data2, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(out_data2, &(out->data[0]), sizeof(float) * out->data.size(), hipMemcpyHostToDevice));

        dim3 blocksPerGrid(graph->indptr.size() - 1, 1, 1);
        dim3 threadsPerBlock(dim, 1, 1);
        gpu_graph_sum_forward<<<blocksPerGrid, threadsPerBlock>>>(in_data2, out_data2, graph_indptr2, graph_indices2, dim);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(&out->data[0], out_data2, sizeof(float) * out->data.size(), hipMemcpyDeviceToHost));  
    }

    call_forward++;
    call_forward %= 2;

    timer_stop(TMR_GRAPHSUM_FW);
}

__global__ void gpu_graph_sum_backward(float *in_grad, float *out_grad, int *graph_indptr, int *graph_indices, const int dim)
{
    int src = blockIdx.x;
    int j = threadIdx.x;

    in_grad[src * dim + j] = 0;

    for (int i = graph_indptr[src]; i < graph_indptr[src + 1]; i++)
    {
        int dst = graph_indices[i];
        float coef = 1.0 / sqrtf(
                               (graph_indptr[src + 1] - graph_indptr[src]) * (graph_indptr[dst + 1] - graph_indptr[dst]));
        in_grad[src * dim + j] += coef * out_grad[dst * dim + j];
    }
}

void GraphSum::backward()
{
    timer_start(TMR_GRAPHSUM_BW);
    if (call_backward == 0)
    {
        CHECK(hipMemcpy(graph_indptr, &(graph->indptr[0]), sizeof(int) * graph->indptr.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(graph_indices, &(graph->indices[0]), sizeof(int) * graph->indices.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(in_grad, &(in->grad[0]), sizeof(float) * in->grad.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(out_grad, &(out->grad[0]), sizeof(float) * out->grad.size(), hipMemcpyHostToDevice));

        dim3 blocksPerGrid(graph->indptr.size() - 1, 1, 1);
        dim3 threadsPerBlock(dim, 1, 1);
        gpu_graph_sum_backward<<<blocksPerGrid, threadsPerBlock>>>(in_grad, out_grad, graph_indptr, graph_indices, dim);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(&in->grad[0], in_grad, sizeof(float) * in->grad.size(), hipMemcpyDeviceToHost));
    } else
    {
        CHECK(hipMemcpy(graph_indptr2, &(graph->indptr[0]), sizeof(int) * graph->indptr.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(graph_indices2, &(graph->indices[0]), sizeof(int) * graph->indices.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(in_grad2, &(in->grad[0]), sizeof(float) * in->grad.size(), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(out_grad2, &(out->grad[0]), sizeof(float) * out->grad.size(), hipMemcpyHostToDevice));

        dim3 blocksPerGrid(graph->indptr.size() - 1, 1, 1);
        dim3 threadsPerBlock(dim, 1, 1);
        gpu_graph_sum_backward<<<blocksPerGrid, threadsPerBlock>>>(in_grad2, out_grad2, graph_indptr2, graph_indices2, dim);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(&in->grad[0], in_grad2, sizeof(float) * in->grad.size(), hipMemcpyDeviceToHost));
    }

    call_backward++;
    call_backward %= 2;
    timer_stop(TMR_GRAPHSUM_BW);
}

// ################################################################################################################

/**
 * Each predicted class probability is compared to the actual class desired and a loss is computed to penalize the proabability based on how far it is with respect to the actual expected value.
 * Also called logaritmic loss. 
*/
CrossEntropyLoss::CrossEntropyLoss(Variable *logits, int *truth, float *loss, int num_classes) :
        logits(logits), truth(truth), loss(loss), num_classes(num_classes) {}

void CrossEntropyLoss::forward(bool training) {
    timer_start(TMR_LOSS_FW);
    float total_loss = 0;
    int count = 0;
    if (training) logits->zero_grad();
    for (int i = 0; i < logits->data.size() / num_classes; i++) {
        if (truth[i] < 0) continue;
        count++;
        float *logit = &logits->data[i * num_classes];
        float max_logit = -1e30, sum_exp = 0;
        for (int j = 0; j < num_classes; j++)
            max_logit = fmax(max_logit, logit[j]);
        for (int j = 0; j < num_classes; j++) {
            logit[j] -= max_logit;
            sum_exp += expf(logit[j]);
        }
        total_loss += logf(sum_exp) - logit[truth[i]];

        if (training) {
            for (int j = 0; j < num_classes; j++) {
                float prob = expf(logit[j]) / sum_exp;
                logits->grad[i * num_classes + j] = prob;
            }
            logits->grad[i * num_classes + truth[i]] -= 1.0;
        }
    }
    *loss = total_loss / count;
    if (training)
        for (float & i : logits->grad)
            i /= count;
    timer_stop(TMR_LOSS_FW);
}

void CrossEntropyLoss::backward() {
}

// ################################################################################################################

float *in_data3, *in_grad3;
bool *mask_gpu;

/**
 * Rectified Linear Unit activation function.
 * If input is negative it will output 0.
 */
ReLU::ReLU(Variable *in)
{
    this->in = in;
    mask = new bool[in->data.size()];
	
	hipMalloc(&in_data3, in->data.size() * sizeof(float));
	hipMalloc(&in_grad3, in->grad.size() * sizeof(float));
	hipMalloc(&mask_gpu, in->data.size() * sizeof(bool));
}

ReLU::~ReLU()
{
    delete[] mask;
}

__global__ void gpu_relu_forward(float *in_data, bool *mask, const bool training){
	int i = blockIdx.x;
	
    bool keep = in_data[i] > 0;
    if (training)
        mask[i] = keep;
    if (!keep)
        in_data[i] = 0;	
}

void ReLU::forward(bool training)
{
    timer_start(TMR_RELU_FW);
	
	CHECK(hipMemcpy(in_data3, &(in->data[0]), sizeof(float) * in->data.size(), hipMemcpyHostToDevice));
	
	dim3 blocksPerGrid(in->data.size(), 1, 1);
    dim3 threadsPerBlock(1, 1, 1);
    gpu_relu_forward<<<blocksPerGrid, threadsPerBlock>>>(in_data3, mask_gpu, training);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
	
	CHECK(hipMemcpy(&in->data[0], in_data3, sizeof(float) * in->data.size(), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(mask, mask_gpu, in->data.size() * sizeof(bool), hipMemcpyDeviceToHost));
	
    timer_stop(TMR_RELU_FW);
}

__global__ void gpu_relu_backward(float *in_grad, bool *mask){
	int i = blockIdx.x;
	
    if (!mask[i])
        in_grad[i] = 0;
}

void ReLU::backward()
{
    timer_start(TMR_RELU_BW);
	
	CHECK(hipMemcpy(in_grad3, &(in->grad[0]), sizeof(float) * in->grad.size(), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(mask_gpu, mask, in->data.size() * sizeof(bool), hipMemcpyHostToDevice));
	
	dim3 blocksPerGrid(in->data.size(), 1, 1);
    dim3 threadsPerBlock(1, 1, 1);
    gpu_relu_backward<<<blocksPerGrid, threadsPerBlock>>>(in_grad3, mask_gpu);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
	
	CHECK(hipMemcpy(&in->grad[0], in_grad3, sizeof(float) * in->grad.size(), hipMemcpyDeviceToHost));

    timer_stop(TMR_RELU_BW);
}

// ################################################################################################################

/**
 * The dropout layer randomly sets input units to 0 with a frequency of P at each step during training time to prevent overfitting.
 * Inputs that are not set to 0 are scaled up by 1/(1-P).
 */
Dropout::Dropout(Variable *in, float p)
{
    this->in = in;
    this->p = p;
    if (!in->grad.empty())
        mask = new int[in->data.size()];
    else
        mask = nullptr;
}

Dropout::~Dropout()
{
    if (mask)
        delete[] mask;
}

void Dropout::forward(bool training)
{
    if (!training)
        return;
    timer_start(TMR_DROPOUT_FW);
    const int threshold = int(p * MY_RAND_MAX);
    float scale = 1 / (1 - p);
    for (int i = 0; i < in->data.size(); i++)
    {
        bool keep = (int)RAND() >= threshold;
        in->data[i] *= keep ? scale : 0;
        if (mask)
            mask[i] = keep;
    }
    timer_stop(TMR_DROPOUT_FW);
}

void Dropout::backward()
{
    if (!mask)
        return;
    timer_start(TMR_DROPOUT_BW);
    float scale = 1 / (1 - p);
    for (int i = 0; i < in->data.size(); i++)
        in->grad[i] *= mask[i] ? scale : 0;
    timer_stop(TMR_DROPOUT_BW);
}

// ################################################################################################################