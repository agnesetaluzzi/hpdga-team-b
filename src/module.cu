#include "hip/hip_runtime.h"
#include "../include/module.h"
#include "../include/rand.h"
#include "../include/timer.h"
#include <vector>

/* error handling for CUDA API functions */
#define CHECK(call)                                                  \
    {                                                                \
        const hipError_t err = call;                                \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

/* Check to kernel call */
#define CHECK_KERNELCALL()                                           \
    {                                                                \
        const hipError_t err = hipGetLastError();                  \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

__global__ void gpu_zero(float *data, int *p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    data[i * (*p) + k] = 0;
}

float *a_gpu, *b_gpu, *c_gpu;
int *m_gpu, *n_gpu, *p_gpu;

// ################################################################################################################
/**
 * Dense matrix multiplication layer.
 */
Matmul::Matmul(Variable *a, Variable *b, Variable *c, int m, int n, int p) : a(a), b(b), c(c), m(m), n(n), p(p) {}

__global__ void gpu_matmul_forward(float *a_gpu, float *b_gpu, float *c_gpu, int *m, int *n, int *p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    c_gpu[i * (*p) + k] = 0;

    for (int j = 0; j < (*n); j++)
        c_gpu[i * (*p) + k] += a_gpu[i * (*n) + j] * b_gpu[j * (*p) + k];
}

void Matmul::forward(bool training)
{
    timer_start(TMR_MATMUL_FW);

    // GPUs do not support std::vector, etc. so I have to use arrays
    // since the spec now guarantees vectors store their elements contiguously, to transform a std::vector to an array:
    // std::vector<double> v;
    // double *a = &v[0];

    /*dim3 blocksPerGrid(m, 1, 1);
    dim3 threadsPerBlock(p, 1, 1);
    gpu_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(a_gpu, b_gpu, c_gpu, m_gpu, n_gpu, p_gpu);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());*/

    c->zero();
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            for (int k = 0; k < p; k++)
                c->data[i * p + k] += a->data[i * n + j] * b->data[j * p + k];
        }

    // convert an array to a std::vector:
    // int src[] = { 1, 2, 3, 4, 5 };
    // int n = sizeof(src) / sizeof(src[0]);
    // std::vector<int> dest(src, src + n);

    /*float *c_data_from_gpu;
    c_data_from_gpu = (float *)malloc(m * p * sizeof(float));
    CHECK(hipMemcpy(c_data_from_gpu, c_gpu, sizeof(float) * m * p, hipMemcpyDeviceToHost));
    std::vector<float> c_data_from_gpu_vector(c_data_from_gpu, c_data_from_gpu + m * p);
    c->data = c_data_from_gpu_vector;

    CHECK(hipFree(a_gpu));
    CHECK(hipFree(b_gpu));
    CHECK(hipFree(c_gpu));
    CHECK(hipFree(m_gpu));
    CHECK(hipFree(n_gpu));
    CHECK(hipFree(p_gpu));*/

    timer_stop(TMR_MATMUL_FW);
}

void Matmul::backward()
{
    timer_start(TMR_MATMUL_BW);
    a->zero_grad();
    b->zero_grad();
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            float tmp = 0;
            for (int k = 0; k < p; k++)
            {
                tmp += c->grad[i * p + k] * b->data[j * p + k];
                b->grad[j * p + k] += c->grad[i * p + k] * a->data[i * n + j];
            }
            a->grad[i * n + j] = tmp;
        }
    timer_stop(TMR_MATMUL_BW);
}

// ################################################################################################################

/**
 * A sparse matrix multiplication layer.
 */
SparseMatmul::SparseMatmul(Variable *a, Variable *b, Variable *c, SparseIndex *sp, int m, int n, int p) : a(a), b(b), c(c), sp(sp), m(m), n(n), p(p) {}

__global__ void gpu_sparse_matmul_forward(float *a_gpu, float *b_gpu, float *c_gpu, int *sp_indptr_gpu, int *sp_indices_gpu, int *p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

    for (int jj = sp_indptr_gpu[i]; jj < sp_indptr_gpu[i + 1]; jj++)
    {
        int j = sp_indices_gpu[jj];
        c_gpu[i * (*p) + k] += a_gpu[jj] * b_gpu[j * (*p) + k];
    }
}

void SparseMatmul::forward(bool training)
{
    timer_start(TMR_SPMATMUL_FW);

    int *sp_indptr_gpu, *sp_indices_gpu;

    CHECK(hipMalloc(&a_gpu, sizeof(float) * m * n));
    CHECK(hipMalloc(&b_gpu, sizeof(float) * n * p));
    CHECK(hipMalloc(&c_gpu, sizeof(float) * m * p));
    CHECK(hipMalloc(&m_gpu, sizeof(int)));
    CHECK(hipMalloc(&n_gpu, sizeof(int)));
    CHECK(hipMalloc(&p_gpu, sizeof(int)));

    CHECK(hipMalloc(&sp_indptr_gpu, sizeof(int) * sp->indptr.size()));
    CHECK(hipMalloc(&sp_indices_gpu, sizeof(int) * sp->indices.size()));

    CHECK(hipMemcpy(a_gpu, &(a->data[0]), sizeof(float) * a->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_gpu, &(b->data[0]), sizeof(float) * b->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(c_gpu, &(c->data[0]), sizeof(float) * c->data.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(m_gpu, &m, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(p_gpu, &p, sizeof(int), hipMemcpyHostToDevice));

    CHECK(hipMemcpy(sp_indptr_gpu, &(sp->indptr[0]), sizeof(int) * sp->indptr.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(sp_indices_gpu, &(sp->indices[0]), sizeof(int) * sp->indices.size(), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(sp->indptr.size() - 1, 1, 1);
    dim3 threadsPerBlock(p, 1, 1);
    dim3 blocksPerGridZero(m, 1, 1);
    gpu_zero<<<blocksPerGridZero, threadsPerBlock>>>(c_gpu, p_gpu);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    gpu_sparse_matmul_forward<<<blocksPerGrid, threadsPerBlock>>>(a_gpu, b_gpu, c_gpu, sp_indptr_gpu, sp_indices_gpu, p_gpu);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());

    /*c->zero();
    for (int i = 0; i < sp->indptr.size() - 1; i++)
        for (int jj = sp->indptr[i]; jj < sp->indptr[i + 1]; jj++)
        {
            int j = sp->indices[jj];
            for (int k = 0; k < p; k++)
                c->data[i * p + k] += a->data[jj] * b->data[j * p + k];
        }*/

    // convert an array to a std::vector:
    // int src[] = { 1, 2, 3, 4, 5 };
    // int n = sizeof(src) / sizeof(src[0]);
    // std::vector<int> dest(src, src + n);

    float *c_data_from_gpu;
    c_data_from_gpu = (float *)malloc(m * p * sizeof(float));
    CHECK(hipMemcpy(c_data_from_gpu, c_gpu, sizeof(float) * m * p, hipMemcpyDeviceToHost));
    std::vector<float> c_data_from_gpu_vector(c_data_from_gpu, c_data_from_gpu + m * p);
    c->data = c_data_from_gpu_vector;

    CHECK(hipFree(a_gpu));
    CHECK(hipFree(b_gpu));
    CHECK(hipFree(c_gpu));
    CHECK(hipFree(m_gpu));
    CHECK(hipFree(n_gpu));
    CHECK(hipFree(p_gpu));
    CHECK(hipFree(sp_indptr_gpu));
    CHECK(hipFree(sp_indices_gpu));

    timer_stop(TMR_SPMATMUL_FW);
}

void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);
    b->zero_grad();
    int row = 0;
    for (int i = 0; i < sp->indptr.size() - 1; i++)
        for (int jj = sp->indptr[i]; jj < sp->indptr[i + 1]; jj++)
        {
            int j = sp->indices[jj];
            for (int k = 0; k < p; k++)
                b->grad[j * p + k] += c->grad[i * p + k] * a->data[jj];
        }
    timer_stop(TMR_SPMATMUL_BW);
}

// ################################################################################################################

/**
 * A specialized sparse matrix multiplication for graphs.
 */
GraphSum::GraphSum(Variable *in, Variable *out, SparseIndex *graph, int dim) : in(in), out(out), graph(graph), dim(dim) {}

void GraphSum::forward(bool training)
{
    timer_start(TMR_GRAPHSUM_FW);
    out->zero();
    for (int src = 0; src < graph->indptr.size() - 1; src++)
        for (int i = graph->indptr[src]; i < graph->indptr[src + 1]; i++)
        {
            int dst = graph->indices[i];
            float coef = 1.0 / sqrtf(
                                   (graph->indptr[src + 1] - graph->indptr[src]) * (graph->indptr[dst + 1] - graph->indptr[dst]));
            for (int j = 0; j < dim; j++)
                // This only works for undirected graphs. Should be out[dst] += coef * in[src]
                out->data[src * dim + j] += coef * in->data[dst * dim + j];
        }
    timer_stop(TMR_GRAPHSUM_FW);
}

void GraphSum::backward()
{
    timer_start(TMR_GRAPHSUM_BW);
    in->zero_grad();
    for (int src = 0; src < graph->indptr.size() - 1; src++)
        for (int i = graph->indptr[src]; i < graph->indptr[src + 1]; i++)
        {
            int dst = graph->indices[i];
            float coef = 1.0 / sqrtf(
                                   (graph->indptr[src + 1] - graph->indptr[src]) * (graph->indptr[dst + 1] - graph->indptr[dst]));
            for (int j = 0; j < dim; j++)
                in->grad[src * dim + j] += coef * out->grad[dst * dim + j];
        }
    timer_stop(TMR_GRAPHSUM_BW);
}

// ################################################################################################################

/**
 * Each predicted class probability is compared to the actual class desired and a loss is computed to penalize the proabability based on how far it is with respect to the actual expected value.
 * Also called logaritmic loss.
 */
CrossEntropyLoss::CrossEntropyLoss(Variable *logits, int *truth, float *loss, int num_classes) : logits(logits), truth(truth), loss(loss), num_classes(num_classes) {}

void CrossEntropyLoss::forward(bool training)
{
    timer_start(TMR_LOSS_FW);
    float total_loss = 0;
    int count = 0;
    if (training)
        logits->zero_grad();
    for (int i = 0; i < logits->data.size() / num_classes; i++)
    {
        if (truth[i] < 0)
            continue;
        count++;
        float *logit = &logits->data[i * num_classes];
        float max_logit = -1e30, sum_exp = 0;
        for (int j = 0; j < num_classes; j++)
            max_logit = fmax(max_logit, logit[j]);
        for (int j = 0; j < num_classes; j++)
        {
            logit[j] -= max_logit;
            sum_exp += expf(logit[j]);
        }
        total_loss += logf(sum_exp) - logit[truth[i]];

        if (training)
        {
            for (int j = 0; j < num_classes; j++)
            {
                float prob = expf(logit[j]) / sum_exp;
                logits->grad[i * num_classes + j] = prob;
            }
            logits->grad[i * num_classes + truth[i]] -= 1.0;
        }
    }
    *loss = total_loss / count;
    if (training)
        for (float &i : logits->grad)
            i /= count;
    timer_stop(TMR_LOSS_FW);
}

void CrossEntropyLoss::backward()
{
}

// ################################################################################################################

/**
 * Rectified Linear Unit activation function.
 * If input is negative it will output 0.
 */
ReLU::ReLU(Variable *in)
{
    this->in = in;
    mask = new bool[in->data.size()];
}

ReLU::~ReLU()
{
    delete[] mask;
}

void ReLU::forward(bool training)
{
    timer_start(TMR_RELU_FW);
    for (int i = 0; i < in->data.size(); i++)
    {
        bool keep = in->data[i] > 0;
        if (training)
            mask[i] = keep;
        if (!keep)
            in->data[i] = 0;
    }
    timer_stop(TMR_RELU_FW);
}

void ReLU::backward()
{
    timer_start(TMR_RELU_BW);
    for (int i = 0; i < in->data.size(); i++)
        if (!mask[i])
            in->grad[i] = 0;
    timer_stop(TMR_RELU_BW);
}

// ################################################################################################################

/**
 * The dropout layer randomly sets input units to 0 with a frequency of P at each step during training time to prevent overfitting.
 * Inputs that are not set to 0 are scaled up by 1/(1-P).
 */
Dropout::Dropout(Variable *in, float p)
{
    this->in = in;
    this->p = p;
    if (!in->grad.empty())
        mask = new int[in->data.size()];
    else
        mask = nullptr;
}

Dropout::~Dropout()
{
    if (mask)
        delete[] mask;
}

void Dropout::forward(bool training)
{
    if (!training)
        return;
    timer_start(TMR_DROPOUT_FW);
    const int threshold = int(p * MY_RAND_MAX);
    float scale = 1 / (1 - p);
    for (int i = 0; i < in->data.size(); i++)
    {
        bool keep = (int)RAND() >= threshold;
        in->data[i] *= keep ? scale : 0;
        if (mask)
            mask[i] = keep;
    }
    timer_stop(TMR_DROPOUT_FW);
}

void Dropout::backward()
{
    if (!mask)
        return;
    timer_start(TMR_DROPOUT_BW);
    float scale = 1 / (1 - p);
    for (int i = 0; i < in->data.size(); i++)
        in->grad[i] *= mask[i] ? scale : 0;
    timer_stop(TMR_DROPOUT_BW);
}

// ################################################################################################################